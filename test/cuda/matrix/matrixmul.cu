#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "kernel.cu"
#include "dev_array.h"
#include <math.h>

#include <chrono>

using namespace std;

int main()
{
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    int N = 1000;
    int SIZE = N*N;

    std::cout << "N: " << N << std::endl;

    // Allocate memory on the host
    vector<float> h_A(SIZE);
    vector<float> h_B(SIZE);
    vector<float> h_C(SIZE);

    // Initialize matrices on the host
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = sin(i);
            h_B[i*N+j] = cos(j);
        }
    }

    auto start = std::chrono::high_resolution_clock::now();

    // Allocate memory on the device
    dev_array<float> d_A(SIZE);
    dev_array<float> d_B(SIZE);
    dev_array<float> d_C(SIZE);

    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);

    const auto dgpu1 = std::chrono::high_resolution_clock::now()-start;
    std::cout << "GPU data init+copy: " << std::chrono::duration_cast<std::chrono::milliseconds>(dgpu1).count() << " ms" << std::endl;

    matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
    hipDeviceSynchronize();

    d_C.get(&h_C[0], SIZE);
    hipDeviceSynchronize();

    const auto dgpu = std::chrono::high_resolution_clock::now()-start;
    std::cout << "GPU total: " << std::chrono::duration_cast<std::chrono::milliseconds>(dgpu).count() << " ms" << std::endl;

    float *cpu_C;
    cpu_C=new float[SIZE];

    // Now do the matrix multiplication on the CPU
    double sum;
    start = std::chrono::high_resolution_clock::now();

    for (size_t row=0; row<N; ++row){
        for (size_t col=0; col<N; ++col){
            sum = 0.0;
            for (size_t n=0; n<N; ++n){
                sum += h_A[row*N+n]*h_B[n*N+col];
            }
            cpu_C[row*N+col] = sum;
        }
    }
    const auto dcpu = std::chrono::high_resolution_clock::now()-start;
    std::cout << "CPU: " << std::chrono::duration_cast<std::chrono::milliseconds>(dcpu).count() << " ms" << std::endl;

    std::cout << "CPU/GPU: " << dcpu/dgpu << std::endl;

    double err = 0.0;
    // Check the result and make sure it is correct
    for (size_t ROW=0; ROW < N; ++ROW) {
        for (size_t COL=0; COL < N; ++COL) {
            const size_t index = ROW*N+COL;
            err += cpu_C[index] - h_C[index];
        }
    }

    cout << "Error: " << err << endl;

    return 0;
}