// Simple cuSparse test:
// create random dense matrix A an identity matrix I on cpu host, copy them to gpu device, convert to sparse CSR format,
// compute C = A*I in CSR sparse format, convert to dense and copy to cpu host, chek C == A
// examples based on https://github.com/NVIDIA/CUDALibrarySamples/tree/master/cuSPARSE
// cuSparse doc: https://docs.nvidia.com/cuda/cusparse/



#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSparseToDense
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

// Row-major order
float* generate_random_dense_matrix(int rows, int cols)
{
    int i, j;
    float *matrix = (float *)malloc(sizeof(float) * rows * cols);
    for (i = 0; i < rows; i++)
    {   
        for (j = 0; j < cols; j++)
        {
            int r = rand();
            matrix[i * cols + j] = 0.0f;
            if (r % 3 == 0)
            {
                matrix[i * cols + j] = rand() % 100;
            }
        }
    }
    return matrix;
}

float* generate_identity_dense_matrix(int rows, int cols)
{
    int i, j;
    float *matrix = (float *)malloc(sizeof(float) * rows * cols);
    for (i = 0; i < rows; i++)
    {   
        for (j = 0; j < cols; j++)
        {
            matrix[i * cols + j] = 0.0f;
            if (i == j)
            {
                matrix[i * cols + j] = 1;
            }
        }
    }
    return matrix;
}

int main(void) {
    // Host problem definition
    int   num_rows   = 1024;
    int   num_cols   = 1024;
    int   ld         = num_cols;    // leading dimension
    int   dense_size = num_cols * num_rows;

    float               alpha       = 1.0f;
    float               beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_32F;

    float* h_dense = generate_random_dense_matrix(num_rows, num_cols);
    float* h_idense = generate_identity_dense_matrix(num_rows, num_cols);
    float* h_bdense = (float *)malloc(sizeof(float) * num_rows * num_cols);

    //--------------------------------------------------------------------------
    // Device memory management
    int   *d_csr_offsets, *d_csr_columns, *d_icsr_offsets, *d_icsr_columns, *d_bcsr_offsets, *d_bcsr_columns;
    float *d_csr_values, *d_icsr_values, *d_bcsr_values,  *d_dense, *d_idense, *d_bdense;

    CHECK_CUDA( hipMalloc((void**) &d_dense, dense_size * sizeof(float)))
    CHECK_CUDA( hipMalloc((void**) &d_idense, dense_size * sizeof(float)))
    CHECK_CUDA( hipMalloc((void**) &d_bdense, dense_size * sizeof(float)))

    CHECK_CUDA( hipMalloc((void**) &d_csr_offsets, (num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &d_icsr_offsets, (num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &d_bcsr_offsets, (num_rows + 1) * sizeof(int)) )
    
    CHECK_CUDA( hipMemcpy(d_dense, h_dense, dense_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(d_idense, h_idense, dense_size * sizeof(float),
                            hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matSA;
    hipsparseSpMatDescr_t matSI;
    hipsparseSpMatDescr_t matSB;
    hipsparseDnMatDescr_t matDA;
    hipsparseDnMatDescr_t matDI;
    hipsparseDnMatDescr_t matDB;
    
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;

    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A, create dense matrix I
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matDA, num_rows, num_cols, ld, d_dense,
                                        computeType, HIPSPARSE_ORDER_ROW) )
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matDI, num_rows, num_cols, ld, d_idense,
                                        computeType, HIPSPARSE_ORDER_ROW) )
    
    // Create sparse matrix A in CSR format, create sparse matrix I in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matSA, num_rows, num_cols, 0,
                                      d_csr_offsets, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, computeType) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matSI, num_rows, num_cols, 0,
                                      d_icsr_offsets, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, computeType) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseDenseToSparse_bufferSize(
                                        handle, matDA, matSA,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize) )

    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute Dense to Sparse conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_analysis(handle, matDA, matSA,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )
    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matSA, &num_rows_tmp, &num_cols_tmp,
                                         &nnz) )

    // allocate CSR column indices and values
    CHECK_CUDA( hipMalloc((void**) &d_csr_columns, nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &d_csr_values,  nnz * sizeof(float)) )
    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE( hipsparseCsrSetPointers(matSA, d_csr_offsets, d_csr_columns,
                                           d_csr_values) )
    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_convert(handle, matDA, matSA,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )

    CHECK_CUDA( hipFree(dBuffer) )
   
    // the same for I
    CHECK_CUSPARSE( hipsparseDenseToSparse_bufferSize(
                                        handle, matDI, matSI,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )
    CHECK_CUSPARSE( hipsparseDenseToSparse_analysis(handle, matDI, matSI,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matSI, &num_rows_tmp, &num_cols_tmp,
                                         &nnz) )
    CHECK_CUDA( hipMalloc((void**) &d_icsr_columns, nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &d_icsr_values,  nnz * sizeof(float)) )
    CHECK_CUSPARSE( hipsparseCsrSetPointers(matSI, d_icsr_offsets, d_icsr_columns,
                                           d_icsr_values) )
    CHECK_CUSPARSE( hipsparseDenseToSparse_convert(handle, matDI, matSI,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )
    CHECK_CUDA( hipFree(dBuffer) )

    
    CHECK_CUSPARSE( hipsparseCreateCsr(&matSB, num_rows,  num_cols, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, computeType) )

    //--------------------------------------------------------------------------
    // SpGEMM Computation
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

    void*  dBuffer2   = NULL;
    size_t bufferSize2 = 0;

    // ask bufferSize bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matSA, matSI, &beta, matSB,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer, bufferSize) )
    // inspect the matrices SA and SI to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matSA, matSI, &beta, matSB,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize, dBuffer) )

    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matSA, matSI, &beta, matSB,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // compute the intermediate product of A * I
    CHECK_CUSPARSE( hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matSA, matSI, &beta, matSB,
                                           computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2) )
    // get matrix B non-zero entries B_nnz1
    int64_t B_num_rows1, B_num_cols1, B_nnz1;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matSB, &B_num_rows1, &B_num_cols1,
                                         &B_nnz1) )
    // allocate matrix B
    CHECK_CUDA( hipMalloc((void**) &d_bcsr_columns, B_nnz1 * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &d_bcsr_values,  B_nnz1 * sizeof(float)) )

    // NOTE: if 'beta' != 0, the values of B must be update after the allocation
    //       of d_bcsr_values, and before the call of cusparseSpGEMM_copy

    // update matB with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matSB, d_bcsr_offsets, d_bcsr_columns, d_bcsr_values) )


    // if beta != 0, cusparseSpGEMM_copy reuses/updates the values of dC_values

    // copy the final products to the matrix B
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matSA, matSI, &beta, matSB,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )



    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dBuffer2) )

    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matDB, num_rows, num_cols, ld, d_bdense,
                                        computeType, HIPSPARSE_ORDER_ROW) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSparseToDense_bufferSize(
                                        handle, matSB, matDB,
                                        HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                        &bufferSize) )

    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseSparseToDense(handle, matSB, matDB,
                                          HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                          dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matDA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matSA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matDI) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matSI) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matDB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matSB) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(h_bdense, d_bdense, dense_size * sizeof(float),
                           hipMemcpyDeviceToHost) )

    int correct = 1;
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            if (h_dense[i * ld + j] != h_bdense[i * ld + j]) {
                correct = 0;
                break;
            }
        }
    }
    if (correct)
        printf("Test PASSED\n");
    else
        printf("Test FAILED: wrong result\n");

    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(d_csr_offsets) )
    CHECK_CUDA( hipFree(d_csr_columns) )
    CHECK_CUDA( hipFree(d_csr_values) )
    CHECK_CUDA( hipFree(d_dense) )
    CHECK_CUDA( hipFree(d_icsr_offsets) )
    CHECK_CUDA( hipFree(d_icsr_columns) )
    CHECK_CUDA( hipFree(d_icsr_values) )
    CHECK_CUDA( hipFree(d_idense) )
    CHECK_CUDA( hipFree(d_bcsr_offsets) )
    CHECK_CUDA( hipFree(d_bcsr_columns) )
    CHECK_CUDA( hipFree(d_bcsr_values) )
    CHECK_CUDA( hipFree(d_bdense) )

    // host dealloc
    free(h_dense);
    free(h_idense);
    free(h_bdense);
    return EXIT_SUCCESS;
}
